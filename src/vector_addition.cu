#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


#define N 10

__global__
void add(int *a, int *b,int *c){
	//get unique thread id
	int tid = blockIdx.x;

	if (tid < N){
		c[tid] = a[tid]+b[tid];
	}
}


int main()
{

	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	//allocate memory on device for the three arrays
	hipMalloc((void**)&dev_a,N*sizeof(int));
	hipMalloc((void**)&dev_b,N*sizeof(int));
	hipMalloc((void**)&dev_c,N*sizeof(int));

	//fill arrays on host
	for(int i = 0 ; i<N;i++){
		a[i]= -i;
		b[i]= i*i;
	}


	//copy arrays to GPU
	hipMemcpy(dev_a,a,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_c,c,N*sizeof(int),hipMemcpyHostToDevice);


	//call cuda kernel
	add<<<N,1>>>(dev_a,dev_b,dev_c);

	//copy c array back to host
	hipMemcpy(c,dev_c, N*sizeof(int), hipMemcpyDeviceToHost);


	//print results
	for(int i=0; i<N;i++){
		printf("%d + %d = %d\n",a[i],b[i],c[i]);
	}

	//free allocated memory on device
	hipFree(dev_a);
	hipFree(dev_a);
	hipFree(dev_a);

	printf("done");
	return EXIT_SUCCESS;
}
